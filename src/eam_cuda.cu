#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/inner_product.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

/*
 Todo:
 1) The surface tension is not calcualted! 
 -> if (SURFTEN==1 && (curstep%savepropfreq)==1) AddnvvtoPtPn(_SR[jpt],rij,rij,-fp);
 2) It only supports MD and relaxation calculations. MC can be impelemented in a similar way.
 */

#include "eam.h"
#include "lock.h"

#define EQIV(a, b) (abs((a)-(b))<1e-15?1:0)

void EAMFrame::cuda_memory_alloc() {
  int size = _NP*allocmultiple;
  gpuErrchk(hipMalloc(&_d_atpe3b,sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_rhotot,sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_rhotot_padding,sizeof(double)*size&_NNM));
  gpuErrchk(hipMalloc(&_d_embf,  sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_embfp, sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_nbst,  sizeof(int)*size));
  /* eam grid allocation */
  gpuErrchk(hipMalloc(&_d_rho,   sizeof(double)*4*NGRID));
  gpuErrchk(hipMalloc(&_d_rhop,  sizeof(double)*4*NGRID));
  gpuErrchk(hipMalloc(&_d_phi,   sizeof(double)*2*NGRID));
  gpuErrchk(hipMalloc(&_d_phip,  sizeof(double)*2*NGRID));
  gpuErrchk(hipMalloc(&_d_phix,   sizeof(double)*NGRID));
  gpuErrchk(hipMalloc(&_d_phipx,  sizeof(double)*NGRID));
  gpuErrchk(hipMalloc(&_d_frho,   sizeof(double)*2*NGRID));
  gpuErrchk(hipMalloc(&_d_frhop,  sizeof(double)*2*NGRID));
  gpuErrchk(hipMalloc(&_d_rho_spline,   sizeof(double)*4*NGRID*4));
  gpuErrchk(hipMalloc(&_d_phi_spline,   sizeof(double)*2*NGRID*4));
  gpuErrchk(hipMalloc(&_d_phix_spline,   sizeof(double)*4*NGRID));
  gpuErrchk(hipMalloc(&_d_frho_spline,   sizeof(double)*2*NGRID*4));
  gpuErrchk(hipMalloc(&_d_rval,          sizeof(double)*NGRID));
  gpuErrchk(hipMalloc(&_d_rhoval,        sizeof(double)*NGRID));
  /* md data allocation */
  gpuErrchk(hipMalloc(&_d_H_element,      sizeof(double)*3*3));
  gpuErrchk(hipMalloc(&_d_VIRIAL_element, sizeof(double)*3*3));
  gpuErrchk(hipMalloc(&_d_EPOT,       sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_EPOT_IND,   sizeof(double)*size));
  gpuErrchk(hipMalloc(&_d_EPOT_IND_padding,   sizeof(double)*size*_NNM));
  gpuErrchk(hipMalloc(&_d_species,    sizeof(int)*size));
  gpuErrchk(hipMalloc(&_d_fixed,      sizeof(int)*size));
  /* size of nindex is obtained from md.cpp:NbrList_init. mx = size, mz = NNM,
     but we don`t need memory for pointers (shft2) since cuda wants a 1d array. 
   */
  gpuErrchk(hipMalloc(&_d_nindex,     sizeof(int)*size*_NNM));
  gpuErrchk(hipMalloc(&_d_nn,         sizeof(int)*size));
  gpuErrchk(hipMalloc(&_d_SR,         sizeof(G_Vector3)*size));
  gpuErrchk(hipMalloc(&_d_F,          sizeof(G_Vector3)*size));
  gpuErrchk(hipMalloc(&_d_F_padding,  sizeof(G_Vector3)*size*_NNM));
  gpuErrchk(hipMalloc(&_d_VIRIAL_IND_element, sizeof(double)*9*size));
  gpuErrchk(hipMalloc(&_d_VIRIAL_IND_element_padding, sizeof(double)*_NNM*9*size));
  gpuErrchk(hipMalloc(&_d_fscalars,   sizeof(double)*10));
  Realloc( fscalars, double, 10);

#ifdef DEBUG_USECUDA
  Realloc(    _h_d_atpe3b,    double, size   );
  Realloc(    _h_d_rhotot,    double, size   );
  Realloc(    _h_d_embf,      double, size   );
  Realloc(    _h_d_embfp,     double, size   );
  Realloc(    _h_d_nbst,      int,    size   );
  Realloc(    _h_d_EPOT_IND,  double, size   );

  /* Alloc runtime cpu data to device */
  int shft1=_NP*_NNM*sizeof(int);
  int shft2=_NP*sizeof(int *);
  char *_h_d_nindex_mem=0;
  if(shft1+shft2==0) return;
  Realloc(_h_d_nindex_mem,char,(shft1+shft2));
  _h_d_nindex=(int **)(_h_d_nindex_mem+shft1);
  for(int i=0;i<_NP;i++)
    _h_d_nindex[i]=(int *)(_h_d_nindex_mem+i*_NNM*sizeof(int));
  Realloc(_h_d_nn,            int,     size    );
  Realloc(_h_d_SR,            Vector3, size    );
  Realloc(_h_d_F,             Vector3, size    );
  Realloc(_h_d_VIRIAL_IND,    Matrix33,size    );
  Realloc(_h_d_fscalars,      double,  10      );
#endif
}

void EAMFrame::cuda_memcpy_all() {
  int size = _NP*allocmultiple;
  gpuErrchk(hipMemcpy(_d_atpe3b,      atpe3b,      sizeof(double)*size,      hipMemcpyHostToDevice) );
  gpuErrchk(hipMemcpy(_d_rhotot,      rhotot,      sizeof(double)*size,      hipMemcpyHostToDevice) );
  gpuErrchk(hipMemcpy(_d_embf,        embf,        sizeof(double)*size,      hipMemcpyHostToDevice) );
  gpuErrchk(hipMemcpy(_d_embfp,       embfp,       sizeof(double)*size,      hipMemcpyHostToDevice) );
  gpuErrchk(hipMemcpy(_d_nbst,        nbst,        sizeof(int)*size,         hipMemcpyHostToDevice) );

  /* copy eam data on grid to device */
  gpuErrchk(hipMemcpy(_d_rho,         rho,         sizeof(double)*4*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_rhop,        rhop,        sizeof(double)*4*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phi,         phi,         sizeof(double)*2*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phip,        phip,        sizeof(double)*2*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phix,        phix,        sizeof(double)*NGRID,     hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phipx,       phipx,       sizeof(double)*NGRID,     hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_frho,        frho,        sizeof(double)*2*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_frhop,       frhop,       sizeof(double)*2*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_rho_spline,  rho_spline,  sizeof(double)*4*NGRID*4, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phi_spline,  phi_spline,  sizeof(double)*2*NGRID*4, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_phix_spline, phix_spline, sizeof(double)*4*NGRID,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_frho_spline, frho_spline, sizeof(double)*2*NGRID*4, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_rval,        rval,        sizeof(double)*NGRID,     hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_rhoval,      rhoval,      sizeof(double)*NGRID,     hipMemcpyHostToDevice));
  
  NbrList_refresh();
  gpuErrchk(hipMemcpy(_d_nindex,   nindex[0],      sizeof(int)*size*_NNM,    hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_nn,          nn,          sizeof(int)*size,         hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_species,     species,     sizeof(int)*size,         hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_SR,          _SR,         sizeof(G_Vector3)*size,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_F,           _F,          sizeof(G_Vector3)*size,   hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_VIRIAL_IND_element, _VIRIAL_IND[0].element,sizeof(double)*9*size, hipMemcpyHostToDevice));
  
  fscalars[0]=rmass  ;
  fscalars[1]=rlatt  ;
  fscalars[2]=drar   ;
  fscalars[3]=drhoar ;
  fscalars[4]=actual ;
  fscalars[5]=actual2;
  fscalars[6]=rmin   ;
  fscalars[7]=petrip ;
  fscalars[8]=rhocon ;
  fscalars[9]=rhomax ;
  gpuErrchk(hipMemcpy(_d_fscalars,    fscalars,    sizeof(double)*10,        hipMemcpyHostToDevice));
}


__device__ double spline(double* _d_spline_coeff,int ind, double qq)
{
  double a, b, c, d, qq2, qq3, f;
  a = _d_spline_coeff[ind*4+0];
  b = _d_spline_coeff[ind*4+1];
  c = _d_spline_coeff[ind*4+2];
  d = _d_spline_coeff[ind*4+3];
  qq2=qq*qq; qq3=qq2*qq;
  f=a+b*qq+c*qq2+d*qq3;
  return f;
}  
__device__ double spline1(double* _d_spline_coeff,int ind, double qq)
{
  double b, c, d, qq2, fp;//, qq3
  //double a = _d_spline_coeff[ind*4+0];
  b = _d_spline_coeff[ind*4+1];
  c = _d_spline_coeff[ind*4+2];
  d = _d_spline_coeff[ind*4+3];
  qq2=qq*qq; //qq3=qq2*qq;
  fp=b+2*c*qq+3*d*qq2;
  return fp;
}


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
__device__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
} 
#endif

 __device__ void atomicAddnvv(double* address, double n,G_Vector3 &a,G_Vector3 &b)
    {
        atomicAdd(address+0,n*a.x*b.x);
        atomicAdd(address+1,n*a.x*b.y);
        atomicAdd(address+2,n*a.x*b.z);
        atomicAdd(address+3,n*a.y*b.x);
        atomicAdd(address+4,n*a.y*b.y);
        atomicAdd(address+5,n*a.y*b.z);
        atomicAdd(address+6,n*a.z*b.x);
        atomicAdd(address+7,n*a.z*b.y);
        atomicAdd(address+8,n*a.z*b.z);
    }

__global__ void kernel_rhoeam_0(int _NP, int _NNM, int eamfiletype, int eamgrid,
                              double *_d_rho,double *_d_rhop,double *_d_phi,double *_d_phip,
                              double *_d_phix,double *_d_phipx,
                              double *_d_frho,double *_d_frhop,
                              double *_d_rho_spline, double *_d_phi_spline,
                              double *_d_phix_spline,double *_d_frho_spline,
                              double *_d_rval,double *_d_rhoval,
                              double *_d_atpe3b,double *_d_rhotot, double *_d_embf, double *_d_embfp,
			      double *_d_rhotot_padding,
			      int    *_d_nbst,
                              double *_d_EPOT,
                              double *_d_H_element,
                              double *_d_VIRIAL_element,
                              double *_d_EPOT_IND,
                              double *_d_EPOT_IND_padding,
                              int *_d_species,
                              int *_d_nindex,
                              int *_d_nn,
                              G_Vector3 *_d_SR,
                              G_Vector3 *_d_F,
                              G_Vector3 *_d_F_padding,
                              double * _d_VIRIAL_IND_element,
                              double * _d_VIRIAL_IND_element_padding,
                              double *_d_fscalars)
{
    int i, j, l, jpt, idx, jdx, ind;
    G_Vector3 sij,rij;
    double r2ij, rmagg, qq, qr;
    double rhoi, rhoj;
    G_Matrix33 _d_H(_d_H_element);

    //double _d_rmass   = _d_fscalars[0];
    //double _d_rlatt   = _d_fscalars[1];
    double _d_drar    = _d_fscalars[2];
    double _d_drhoar  = _d_fscalars[3];
    //double _d_actual  = _d_fscalars[4];
    double _d_actual2 = _d_fscalars[5];
    double _d_rmin    = _d_fscalars[6];
    double _d_petrip  = _d_fscalars[7];
    double _d_rhocon  = _d_fscalars[8];
    double _d_rhomax  = _d_fscalars[9];

        /*INFO("rhoeam");*/
        _d_petrip = 0.0;
        //rhocon = 1e-10;
        _d_rhocon = 0.0;            /* replaced by Keonwook Kang, Apr 29, 2011 */
        _d_rhomax = eamgrid*_d_drhoar; 

    for(i=blockDim.x*blockIdx.x+threadIdx.x;i<_NP;i+=blockDim.x*gridDim.x)
    {
        _d_atpe3b[i]=0;
        _d_rhotot[i]=0;
        _d_nbst[i]=0;
     }

    for(i=blockDim.x*blockIdx.x+threadIdx.x;i<_NP;i+=blockDim.x*gridDim.x)
    {
        _d_F[i].clear(); _d_EPOT_IND[i]=0;
	for(l = 0; l<9; l++) {
	  _d_VIRIAL_IND_element[i*9+l] = 0; 
	  _d_VIRIAL_IND_element_padding[i*9+l] = 0;
	 }
        _d_EPOT[i]=0;
    }
}


__global__ void kernel_rhoeam_1(int _NP, int _NNM, int eamfiletype, int eamgrid,
                              double *_d_rho,double *_d_rhop,double *_d_phi,double *_d_phip,
                              double *_d_phix,double *_d_phipx,
                              double *_d_frho,double *_d_frhop,
                              double *_d_rho_spline, double *_d_phi_spline,
                              double *_d_phix_spline,double *_d_frho_spline,
                              double *_d_rval,double *_d_rhoval,
                              double *_d_atpe3b,double *_d_rhotot, double *_d_embf, double *_d_embfp,
			      double *_d_rhotot_padding,
			      int    *_d_nbst,
                              double *_d_EPOT,
                              double *_d_H_element,
                              double *_d_VIRIAL_element,
                              double *_d_EPOT_IND,
                              double *_d_EPOT_IND_padding,
                              int *_d_species,
                              int *_d_nindex,
                              int *_d_nn,
                              G_Vector3 *_d_SR,
                              G_Vector3 *_d_F,
                              G_Vector3 *_d_F_padding,
                              double * _d_VIRIAL_IND_element,
                              double * _d_VIRIAL_IND_element_padding,
                              double *_d_fscalars)
{
    int i, j, l, jpt, idx, jdx, ind;
    G_Vector3 sij,rij;
    double r2ij, rmagg, qq, qr;
    double rhoi, rhoj;
    G_Matrix33 _d_H(_d_H_element);

    //double _d_rmass   = _d_fscalars[0];
    //double _d_rlatt   = _d_fscalars[1];
    double _d_drar    = _d_fscalars[2];
    double _d_drhoar  = _d_fscalars[3];
    //double _d_actual  = _d_fscalars[4];
    double _d_actual2 = _d_fscalars[5];
    double _d_rmin    = _d_fscalars[6];
    double _d_petrip  = _d_fscalars[7];
    double _d_rhocon  = _d_fscalars[8];
    double _d_rhomax  = _d_fscalars[9];

        /*INFO("rhoeam");*/
        _d_petrip = 0.0;
        //rhocon = 1e-10;
        _d_rhocon = 0.0;            /* replaced by Keonwook Kang, Apr 29, 2011 */
        _d_rhomax = eamgrid*_d_drhoar; 

    for(i=blockDim.x*blockIdx.x+threadIdx.x;i<_NP;i+=blockDim.x*gridDim.x)
    {
    
        /* modify here for binary systems (0/1) */
        idx = _d_species[i]; /* type of atom (i) */
        /* do on j-particles */
        for(j=0;j<_d_nn[i];j++)
        {
            /* modify here for binary systems (0/1) */
            jpt=_d_nindex[i*_NNM+j];
            jdx = _d_species[jpt]; /* type of atom (j) */
            if(i>=jpt) continue;
            sij=_d_SR[jpt]-_d_SR[i];
            sij.subint();
            rij=_d_H*sij;
            r2ij=rij.norm2();
            if(r2ij>_d_actual2) continue;
            rmagg=sqrt(r2ij)-_d_rmin;

            _d_nbst[i]++;
            ind = (int)(rmagg/_d_drar);

            if(ind>=eamgrid)
            {
                ind=eamgrid-1;
                printf("ind = %d r=%f in RHOEAM\n",ind, rmagg+_d_rmin);
            }
            else if(ind<0)
            {
                ind=0;
                printf("ind = %d r=%f in RHOEAM\n",ind, rmagg+_d_rmin);
            }
            qq=rmagg-_d_rval[ind];

            if(idx==jdx)
            {
#ifndef _CUBICSPLINE
            rhoi=_d_rho[jdx*NGRID+ind]+ qq*_d_rhop[jdx*NGRID+ind];
#else
            //rhoi = interp(rho[jdx],rhop[jdx],drar,ind,qq);
            rhoi = spline(_d_rho_spline[jdx],ind,qq);
#endif
	    atomicAdd(_d_rhotot+i, rhoi);
	    atomicAdd(_d_rhotot+jpt, rhoi);
            }
            else
            {
              if (eamfiletype == 2)
              {
#ifndef _CUBICSPLINE
              rhoi=_d_rho[jdx*NGRID+ind]+ qq*_d_rhop[jdx*NGRID+ind];
              rhoj=_d_rho[idx*NGRID+ind]+ qq*_d_rhop[idx*NGRID+ind];
#else
              //rhoi = interp(rho[jdx],rhop[jdx],drar,ind,qq);
              //rhoj = interp(rho[idx],rhop[idx],drar,ind,qq);
              rhoi = spline(_d_rho_spline[jdx*NGRID*4],ind,qq);
              rhoj = spline(_d_rho_spline[idx*NGRID*4],ind,qq);
#endif
              } else if (eamfiletype == 4)
              {
#ifndef _CUBICSPLINE
              rhoi=_d_rho[(idx+2)*NGRID+ind]+ qq*_d_rhop[(idx+2)*NGRID+ind];
              rhoj=_d_rho[(jdx+2)*NGRID+ind]+ qq*_d_rhop[(jdx+2)*NGRID+ind];
#else
              rhoi = spline(_d_rho_spline[(idx+2)*NGRID*4],ind,qq);
              rhoj = spline(_d_rho_spline[(jdx+2)*NGRID*4],ind,qq);
#endif
              }
	      atomicAdd(_d_rhotot+i, rhoi);
	      atomicAdd(_d_rhotot+jpt, rhoj);
            }
        }
    }

}


__global__ void kernel_rhoeam_2(int _NP, int _NNM, int eamfiletype, int eamgrid,
                              double *_d_rho,double *_d_rhop,double *_d_phi,double *_d_phip,
                              double *_d_phix,double *_d_phipx,
                              double *_d_frho,double *_d_frhop,
                              double *_d_rho_spline, double *_d_phi_spline,
                              double *_d_phix_spline,double *_d_frho_spline,
                              double *_d_rval,double *_d_rhoval,
                              double *_d_atpe3b,double *_d_rhotot, double *_d_embf, double *_d_embfp,
			      double *_d_rhotot_padding,
			      int    *_d_nbst,
                              double *_d_EPOT,
                              double *_d_H_element,
                              double *_d_VIRIAL_element,
                              double *_d_EPOT_IND,
                              double *_d_EPOT_IND_padding,
                              int *_d_species,
                              int *_d_nindex,
                              int *_d_nn,
                              G_Vector3 *_d_SR,
                              G_Vector3 *_d_F,
                              G_Vector3 *_d_F_padding,
                              double * _d_VIRIAL_IND_element,
                              double * _d_VIRIAL_IND_element_padding,
                              double *_d_fscalars)
{
    int i, j, l, jpt, idx, jdx, ind;
    G_Vector3 sij,rij;
    double r2ij, rmagg, qq, qr;
    double rhoi, rhoj;
    G_Matrix33 _d_H(_d_H_element);

    //double _d_rmass   = _d_fscalars[0];
    //double _d_rlatt   = _d_fscalars[1];
    double _d_drar    = _d_fscalars[2];
    double _d_drhoar  = _d_fscalars[3];
    //double _d_actual  = _d_fscalars[4];
    double _d_actual2 = _d_fscalars[5];
    double _d_rmin    = _d_fscalars[6];
    double _d_petrip  = _d_fscalars[7];
    double _d_rhocon  = _d_fscalars[8];
    double _d_rhomax  = _d_fscalars[9];

        /*INFO("rhoeam");*/
        _d_petrip = 0.0;
        //rhocon = 1e-10;
        _d_rhocon = 0.0;            /* replaced by Keonwook Kang, Apr 29, 2011 */
        _d_rhomax = eamgrid*_d_drhoar; 


    for(i=blockDim.x*blockIdx.x+threadIdx.x;i<_NP;i+=blockDim.x*gridDim.x)
    {
        /* modify here for binary systems (0/1) */
        idx = _d_species[i]; /* type of atom (i) */
        if(_d_rhotot[i]<_d_rhocon)
        {
            _d_rhotot[i]=_d_rhocon;
        }
        if(_d_rhotot[i]>_d_rhomax)
        {
            _d_rhotot[i]=_d_rhomax;
        }
        ind = (int)(_d_rhotot[i]/_d_drhoar);
        if(ind>=eamgrid-1) ind=eamgrid-1;
        qr = _d_rhotot[i] - _d_rhoval[ind];

#ifndef _CUBICSPLINE
        _d_embf[i] = _d_frho[idx*NGRID+ind] + qr*_d_frhop[idx*NGRID+ind];
        _d_embfp[i] = _d_frhop[idx*NGRID+ind] +
          qr*(_d_frhop[idx*NGRID+ind+1]-_d_frhop[idx*NGRID+ind])/_d_drhoar;
#else
        //embf[i] = interp(frho[idx],frhop[idx],drhoar,ind,qr);
        //embfp[i] = interp1(frho[idx],frhop[idx],drhoar,ind,qr);
        _d_embf[i] = spline(_d_frho_spline[idx*4*NGRID],ind,qr);
        _d_embfp[i] = spline1(_d_frho_spline[idx*4*NGRID],ind,qr);
#endif
	if (i <= 3)
	printf("i = %d, _d_embf[i]= %e, _d_embfp[i]=%e\n", i, _d_embf[i], _d_embfp[i]);

        _d_atpe3b[i] = _d_embf[i];
        _d_EPOT_IND[i]+=_d_atpe3b[i];
        _d_EPOT[i]+=_d_atpe3b[i];
    }    
}




void EAMFrame::rhoeam_cuda() {
  gpuErrchk(hipMemcpy(_d_SR,_SR,_NP*sizeof(G_Vector3), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(_d_H_element,_H.element,9*sizeof(double),hipMemcpyHostToDevice));
  _EPOT = 0;
  _VIRIAL.clear();

#ifdef DEBUG_USECUDA
//  assert(check_host_device_memory_transfer() == 0);
#endif
  kernel_rhoeam_0<<< (_NP+31)/32,32 >>>(_NP, _NNM, eamfiletype, eamgrid,
  //kernel_rhoeam_1<<< 1,1 >>>(_NP, _NNM, eamfiletype, eamgrid,
                           _d_rho, _d_rhop, _d_phi, _d_phip,
                           _d_phix,_d_phipx,
                           _d_frho,_d_frhop,
                           _d_rho_spline, _d_phi_spline,
                           _d_phix_spline,_d_frho_spline,
                           _d_rval,_d_rhoval,
                           _d_atpe3b,_d_rhotot, _d_embf,_d_embfp,
			   _d_rhotot_padding,
			   _d_nbst,
                           _d_EPOT,
                           _d_H_element,
                           _d_VIRIAL_element,
                           _d_EPOT_IND,
			   _d_EPOT_IND_padding,
                           _d_species,
                           _d_nindex,
                           _d_nn,
                           _d_SR,
                           _d_F,
                           _d_F_padding,
                           _d_VIRIAL_IND_element,
                           _d_VIRIAL_IND_element_padding,
                           _d_fscalars);

  kernel_rhoeam_1<<< (_NP+31)/32,32 >>>(_NP, _NNM, eamfiletype, eamgrid,
  //kernel_rhoeam_1<<< 1,1 >>>(_NP, _NNM, eamfiletype, eamgrid,
                           _d_rho, _d_rhop, _d_phi, _d_phip,
                           _d_phix,_d_phipx,
                           _d_frho,_d_frhop,
                           _d_rho_spline, _d_phi_spline,
                           _d_phix_spline,_d_frho_spline,
                           _d_rval,_d_rhoval,
                           _d_atpe3b,_d_rhotot, _d_embf,_d_embfp,
			   _d_rhotot_padding,
			   _d_nbst,
                           _d_EPOT,
                           _d_H_element,
                           _d_VIRIAL_element,
                           _d_EPOT_IND,
			   _d_EPOT_IND_padding,
                           _d_species,
                           _d_nindex,
                           _d_nn,
                           _d_SR,
                           _d_F,
                           _d_F_padding,
                           _d_VIRIAL_IND_element,
                           _d_VIRIAL_IND_element_padding,
                           _d_fscalars);
/* debug */
#if 0 //def DEBUG_USECUDA
  gpuErrchk(hipMemcpy(_h_d_rhotot,_d_rhotot, _NP*sizeof(double), hipMemcpyDeviceToHost));
  for(int i = 0;i<_NP;i++)
    printf("atom[%d] rhotot=%e\n",i,_h_d_rhotot[i]);
#endif

  kernel_rhoeam_2<<< (_NP+31)/32,32 >>>(_NP, _NNM, eamfiletype, eamgrid,
  //kernel_rhoeam_2<<< 1,1 >>>(_NP, _NNM, eamfiletype, eamgrid,
                           _d_rho, _d_rhop, _d_phi, _d_phip,
                           _d_phix,_d_phipx,
                           _d_frho,_d_frhop,
                           _d_rho_spline, _d_phi_spline,
                           _d_phix_spline,_d_frho_spline,
                           _d_rval,_d_rhoval,
                           _d_atpe3b,_d_rhotot, _d_embf,_d_embfp,
			   _d_rhotot_padding,
			   _d_nbst,
                           _d_EPOT,
                           _d_H_element,
                           _d_VIRIAL_element,
                           _d_EPOT_IND,
			   _d_EPOT_IND_padding,
                           _d_species,
                           _d_nindex,
                           _d_nn,
                           _d_SR,
                           _d_F,
                           _d_F_padding,
                           _d_VIRIAL_IND_element,
                           _d_VIRIAL_IND_element_padding,
                           _d_fscalars);

    /* debug */
#if 0
  gpuErrchk(hipMemcpy(_h_d_embf,_d_embf, _NP*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_EPOT_IND,_d_EPOT_IND, _NP*sizeof(double), hipMemcpyDeviceToHost));
  for (int i = 0;i<_NP;i++)
        printf("atom[%d] embf=%e, _d_EPOT=%e\n",i,_h_d_embf[i], _h_d_EPOT_IND[i]);
#endif

#if 0
#if 1
  double *_h_EPOT = 0;        /* used for host reduction only */
  Realloc( _h_EPOT,          double,     _NP);
  gpuErrchk(hipMemcpy(_h_EPOT,_d_EPOT, _NP*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_VIRIAL_IND[0].element,_d_VIRIAL_IND_element, _NP*9*sizeof(double), hipMemcpyDeviceToHost));
  for (int i = 0; i<_NP; i++) {// printf("_h_EPOT[%d]= %g\n", i,_h_EPOT[i]);
	  _EPOT += _h_EPOT[i]; } 
  for (int i = 0; i<_NP; i++) { 
	  _VIRIAL += _VIRIAL_IND[i];
  }
  /* Copy force (Vector3 *) back to CPU for relax function to call */ 
  hipMemcpy(_F, _d_F, _NP*sizeof(G_Vector3), hipMemcpyDeviceToHost);
#else
  thrust::device_ptr<double> t_EPOT = thrust::device_pointer_cast(_d_EPOT);
  _EPOT += thrust::reduce(t_EPOT,t_EPOT+_NP); 
#endif
  INFO_Printf("I am here 2.0, _EPOT = %g \n", _EPOT);
#endif

}

__global__ void kernel_assemble_back_force(int _NP, int _NNM, int *_d_nn,int *_d_nindex,G_Vector3 *_d_F,G_Vector3 *_d_F_padding, double* _d_VIRIAL_IND_element, double* _d_VIRIAL_IND_element_padding) { 
  int i, j, jpt, l, m;
  for(i = blockDim.x * blockIdx.x + threadIdx.x; i<_NP;i+=blockDim.x*gridDim.x) {
    for (j = 0; j<_d_nn[i]; j++) { 
      jpt = i*_NNM+j;
      //k=_inv_d_nindex[m][n]: the n_th neighbor of m_th atom is at k_th location of _d_F_padding
      _d_F[i] += _d_F_padding[ _d_nindex[i*_NNM+j] ];
      for(l = 0;l<3;l++) for(m= 0;m<3;m++)
        _d_VIRIAL_IND_element[i*9+l*3+m] += _d_VIRIAL_IND_element_padding[jpt*9+l*3+m];
    }
  }
}


__global__ void kernel_kraeam(int _NP, int _NNM, int eamfiletype, int eamgrid,
                              double *_d_rho, double *_d_rhop, double *_d_phi, double *_d_phip,
                              double *_d_phix, double *_d_phipx,
                              double *_d_frho, double *_d_frhop,
                              double *_d_rho_spline, double *_d_phi_spline,
                              double *_d_phix_spline, double *_d_frho_spline,
                              double *_d_rval, double *_d_rhoval,
                              double *_d_atpe3b,double *_d_rhotot, double *_d_embf, double *_d_embfp,
			      int    *_d_nbst,
                              double *_d_EPOT,
                              double *_d_H_element,
                              double *_d_VIRIAL_element,
                              double *_d_EPOT_IND,
                              int *_d_species,
                              int *_d_nindex,
                              int *_d_nn,
                              G_Vector3 *_d_SR,
                              G_Vector3 *_d_F,
                              G_Vector3 *_d_F_padding,
                              double *_d_VIRIAL_IND_element,
                              double *_d_VIRIAL_IND_element_padding,
                              double *_d_fscalars)
{
    int i, j, l, m, jpt, idx, jdx, ind;
    G_Vector3 sij,rij,fij;
    G_Matrix33 _d_H(_d_H_element);
    G_Matrix33 _d_VIRIAL(_d_VIRIAL_element);
    double r2ij, rmagg, pp, qq, fcp, fpp, fp, denspi, denspj;

    //double _d_rmass   = _d_fscalars[0];
    //double _d_rlatt   = _d_fscalars[1];
    double _d_drar    = _d_fscalars[2];
    //double _d_drhoar  = _d_fscalars[3];
    //double _d_actual  = _d_fscalars[4];
    double _d_actual2 = _d_fscalars[5];
    double _d_rmin    = _d_fscalars[6];
    //double _d_petrip  = _d_fscalars[7];
    //double _d_rhocon  = _d_fscalars[8];
    //double _d_rhomax  = _d_fscalars[9];
    
    /*	start calculation
     *  calculate pair contribution to total energy and forces
     */
    
    /* do on i-particles */
    for(i=blockDim.x*blockIdx.x+threadIdx.x;i<_NP;i+=blockDim.x*gridDim.x)
    {
        /* modify here for binary systems (0/1) */
        idx = _d_species[i]; /* type of atom (i) */
        /* do on j-particles */
        for(j=0;j<_d_nn[i];j++)
        {
            /* modify here for binary systems (0/1) */
            jpt=_d_nindex[i*_NNM+j];
            jdx = _d_species[jpt]; /* type of atom (j) */
            if(i>=jpt) continue;
            sij=_d_SR[jpt]-_d_SR[i];
            sij.subint();
            rij=_d_H*sij;
            r2ij=rij.norm2();
            if(r2ij>_d_actual2) continue;
            rmagg=sqrt(r2ij)-_d_rmin;

            ind = int(rmagg/_d_drar);
            
            if(ind>=eamgrid)
            {
                ind=eamgrid-1;
                printf("ind = %d in RHOEAM\n",ind);
            }
            else if(ind<0)
            {
                ind=0;
                printf("ind = %d in RHOEAM\n",ind);
            }
            qq=rmagg-_d_rval[ind];

            if(idx==jdx)
            {
#ifndef _CUBICSPLINE
              pp = _d_phi[idx*NGRID+ind] + qq*_d_phip[idx*NGRID+ind];
              fpp = _d_phip[idx*NGRID+ind] +
                qq*(_d_phip[idx*NGRID+ind+1]-_d_phip[idx*NGRID+ind])/_d_drar;
#else
              //pp = interp(phi[idx],phip[idx],drar,ind,qq);
              //fpp = interp1(phi[idx],phip[idx],drar,ind,qq);
              pp = spline(_d_phi_spline[idx*NGRID*4],ind,qq);
              fpp = spline1(_d_phi_spline[idx*NGRID*4],ind,qq);
#endif
            }
            else
            {
#ifndef _CUBICSPLINE
              pp =_d_phix[ind] + qq*_d_phipx[ind];
              fpp = _d_phipx[ind] + qq*(_d_phipx[ind+1]-_d_phipx[ind])/_d_drar;
#else
              //pp = interp(phix,phipx,drar,ind,qq);
              //fpp = interp1(phix,phipx,drar,ind,qq);
              pp = spline(_d_phix_spline,ind,qq);
              fpp = spline1(_d_phix_spline,ind,qq);
#endif
            }
            //INFO_Printf("phi = %20.18e\n",pp);
            if ( (idx==jdx) || (eamfiletype==2) )
            {
#ifndef _CUBICSPLINE
            denspi = _d_rhop[idx*NGRID+ind] +
                qq*(_d_rhop[idx*NGRID+ind+1]-_d_rhop[idx*NGRID+ind])/_d_drar ;
            denspj = _d_rhop[jdx*NGRID+ind] +
                qq*(_d_rhop[jdx*NGRID+ind+1]-_d_rhop[jdx*NGRID+ind])/_d_drar ; /* typo idx fixed to jdx ! */
#else
            //denspi = interp1(rho[idx],rhop[idx],drar,ind,qq);
            //denspj = interp1(rho[jdx],rhop[jdx],drar,ind,qq);
            denspi = spline1(_d_rho_spline[idx],ind,qq);
            denspj = spline1(_d_rho_spline[jdx],ind,qq);
#endif
            } else if ( (idx!=jdx) && (eamfiletype==4) ) {
#ifndef _CUBICSPLINE
            denspi = _d_rhop[(jdx+2)*NGRID+ind] +
                qq*(_d_rhop[(jdx+2)*NGRID+ind+1]-_d_rhop[(jdx+2)*NGRID+ind])/_d_drar ;
            denspj = _d_rhop[(idx+2)*NGRID+ind] +
                qq*(_d_rhop[(idx+2)*NGRID+ind+1]-_d_rhop[(idx+2)*NGRID+ind])/_d_drar ;
#else
            denspi = spline1(_d_rho_spline[(jdx+2)*NGRID*4],ind,qq);
            denspj = spline1(_d_rho_spline[(idx+2)*NGRID*4],ind,qq);
#endif
            }

            fcp = denspj * _d_embfp[i] + denspi * _d_embfp[jpt];
            fp = (fpp + fcp) / (rmagg+_d_rmin);
            
	    atomicAdd(_d_EPOT_IND+i, 0.5*pp);
	    atomicAdd(_d_EPOT_IND+jpt, 0.5*pp);
	    atomicAdd(_d_EPOT+i, pp);
            
            fij=rij*fp;
//????????????????????????????????????????????????
	    atomicAdd(&(_d_F[i].x),fij[0]);
	    atomicAdd(&(_d_F[i].y),fij[1]);
	    atomicAdd(&(_d_F[i].z),fij[2]);
	    atomicAdd(&(_d_F[jpt].x),-fij[0]);
	    atomicAdd(&(_d_F[jpt].y),-fij[1]);
	    atomicAdd(&(_d_F[jpt].z),-fij[2]);

//????????????????????????????????????????????????
	    atomicAddnvv(_d_VIRIAL_IND_element+i*9,-.5*fp,rij,rij);
	    atomicAddnvv(_d_VIRIAL_IND_element+jpt*9,-.5*fp,rij,rij);

            //_VIRIAL.addnvv(-fp,rij,rij);
            //assert(SURFTEN==0);
#if 0
            if (SURFTEN==1 && (curstep%savepropfreq)==1) AddnvvtoPtPn(_SR[jpt],rij,rij,-fp);
#endif
        }
    }
}

void EAMFrame::kraeam_cuda() {
#if 1
  //kernel_kraeam<<< (_NP+31)/32,32 >>>(_NP, _NNM, eamfiletype, eamgrid,
  kernel_kraeam<<< 1,1 >>>(_NP,  _NNM, eamfiletype,  eamgrid,
                           _d_rho, _d_rhop, _d_phi, _d_phip,
                           _d_phix, _d_phipx,
                           _d_frho, _d_frhop,
                           _d_rho_spline, _d_phi_spline,
                           _d_phix_spline, _d_frho_spline,
                           _d_rval, _d_rhoval,
                           _d_atpe3b, _d_rhotot, _d_embf, _d_embfp,
			   _d_nbst,
                           _d_EPOT,
                           _d_H_element,
                           _d_VIRIAL_element,
                           _d_EPOT_IND,
                           _d_species,
                           _d_nindex,
                           _d_nn,
                           _d_SR,
                           _d_F,
                           _d_F_padding,
                           _d_VIRIAL_IND_element,
                           _d_VIRIAL_IND_element_padding,
                           _d_fscalars);

//  kernel_assemble_back_force<<<1,1>>>(_NP, _NNM, _d_nn, _d_nindex, _d_F, _d_F_padding, _d_VIRIAL_IND_element, _d_VIRIAL_IND_element_padding);
#if 1
  double *_h_EPOT = 0;        /* used for host reduction only */
  Realloc( _h_EPOT,          double,     _NP);
  gpuErrchk(hipMemcpy(_EPOT_IND,_d_EPOT_IND, _NP*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_VIRIAL_IND[0].element,_d_VIRIAL_IND_element, _NP*9*sizeof(double), hipMemcpyDeviceToHost));
  for (int i = 0; i<_NP; i++) { //printf("_h_EPOT[%d]= %g\n", i,_h_EPOT[i]);
	  _EPOT += _EPOT_IND[i]; } 
  for (int i = 0; i<_NP; i++) { 
	  _VIRIAL += _VIRIAL_IND[i];
  }
  /* Copy force (Vector3 *) back to CPU for relax function to call */ 
  hipMemcpy(_F, _d_F, _NP*sizeof(G_Vector3), hipMemcpyDeviceToHost);
  for(int i=0;i<_NP;i++)
    {
        INFO_Printf("atom[%d] _F=%e,%e,%e, _EPOT_IND=%e, _EPOT=%e\n",i,_F[i].x, _F[i].y, _F[i].z, _EPOT_IND[i], _EPOT);
    }

#else
  thrust::device_ptr<double> t_EPOT = thrust::device_pointer_cast(_d_EPOT);
  INFO_Printf("I am here 5.0, _EPOT = %g \n", _EPOT);
  _EPOT += thrust::reduce(t_EPOT,t_EPOT+_NP); 
#endif
  INFO_Printf("I am here 2, _EPOT = %g \n", _EPOT);
#endif
}

#ifdef DEBUG_USECUDA
int EAMFrame::check_host_device_memory_transfer() 
{
  INFO_Printf("I am in check_host_device memory transfer\n");
  assert(sizeof(G_Vector3) == sizeof(Vector3));
  assert(sizeof(G_Matrix33) == sizeof(Matrix33));
  assert(_NP>0); assert(_NNM > 0);
  assert(_H[0][0]>0 && _H[1][1]>0 && _H[2][2]>0);

  int size = _NP*allocmultiple;

  gpuErrchk(hipMemcpy(    _h_d_atpe3b,    _d_atpe3b,      sizeof(double)*size,      hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(    _h_d_rhotot,    _d_rhotot,      sizeof(double)*size,      hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(    _h_d_embf,      _d_embf,        sizeof(double)*size,      hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(    _h_d_embfp,     _d_embfp,       sizeof(double)*size,      hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(    _h_d_nbst,      _d_nbst,        sizeof(int)*size,         hipMemcpyDeviceToHost));

  /* copy eam data on grid to device */
  gpuErrchk(hipMemcpy( _h_d_rho,         _d_rho,         sizeof(double)*4*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_rhop,        _d_rhop,        sizeof(double)*4*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phi,         _d_phi,         sizeof(double)*2*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phip,        _d_phip,        sizeof(double)*2*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phix,        _d_phix,        sizeof(double)*NGRID,     hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phipx,       _d_phipx,       sizeof(double)*NGRID,     hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_frho,        _d_frho,        sizeof(double)*2*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_frhop,       _d_frhop,       sizeof(double)*2*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_rho_spline,  _d_rho_spline,  sizeof(double)*4*NGRID*4, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phi_spline,  _d_phi_spline,  sizeof(double)*2*NGRID*4, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_phix_spline, _d_phix_spline, sizeof(double)*4*NGRID,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_frho_spline, _d_frho_spline, sizeof(double)*2*NGRID*4, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_rval,        _d_rval,        sizeof(double)*NGRID,     hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy( _h_d_rhoval,      _d_rhoval,      sizeof(double)*NGRID,     hipMemcpyDeviceToHost));

  /* initial copy of runtime cpu data to device */
  gpuErrchk(hipMemcpy(_h_d_nindex[0],  _d_nindex,          sizeof(int)*size*_NNM,    hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_nn,      _d_nn,              sizeof(int)*size,         hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_SR,      _d_SR,              sizeof(G_Vector3)*size,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_F,       _d_F,               sizeof(G_Vector3)*size,   hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_VIRIAL_IND[0].element, _d_VIRIAL_IND_element, sizeof(double)*9*size, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(_h_d_fscalars,  _d_fscalars,      sizeof(double)*10,        hipMemcpyDeviceToHost));

  for (int i = 0;i<size;i++)   assert(EQIV(atpe3b[i], _h_d_atpe3b[i]));
  for (int i = 0;i<size;i++)   assert(EQIV(rhotot[i], _h_d_rhotot[i]));
  for (int i = 0;i<size;i++)   assert(EQIV(embf[i],   _h_d_embf[i]  ));
  for (int i = 0;i<size;i++)   assert(EQIV(embfp[i],  _h_d_embfp[i] ));
  for (int i = 0;i<size;i++)   assert(EQIV(nbst[i],   _h_d_nbst[i]  ));
  for (int i = 0;i<2; i++) for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(rho[i][j],  _h_d_rho[i][j]));
  for (int i = 0;i<2;i++)  for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(rhop[i][j], _h_d_rhop[i][j]));
  for (int i = 0;i<2;i++)  for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(phi[i][j], _h_d_phi[i][j]));
  for (int i = 0;i<2;i++)  for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(phip[i][j], _h_d_phip[i][j]));
  for (int i = 0;i<NGRID;i++) 
	  assert(EQIV(phix[i], _h_d_phix[i]));
  for (int i = 0;i<NGRID;i++)  
	  assert(EQIV(phipx[i], _h_d_phipx[i]));
  for (int i = 0;i<2;i++)  for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(frho[i][j], _h_d_frho[i][j]));
  for (int i = 0;i<2;i++)  for (int j = 0;j<NGRID;j++) 
	  assert(EQIV(frhop[i][j], _h_d_frhop[i][j]));
  for (int i = 0;i<2;i++) for(int j = 0;j<NGRID;j++) for(int k= 0;k<4;k++) 
	  assert(EQIV(rho_spline[i][j][k], _h_d_rho_spline[i][j][k]));
  for (int i = 0;i<2;i++) for(int j = 0;j<NGRID;j++) for(int k= 0;k<4;k++)
       	  assert(EQIV(phi_spline[i][j][k], _h_d_phi_spline[i][j][k]));
  for (int i = 0;i<NGRID;i++) for (int j = 0;j<4;j++) 
	  assert(EQIV(phix_spline[i][j], _h_d_phix_spline[i][j]));
  for (int i = 0;i<2;i++) for(int j = 0;j<NGRID;j++) for(int k= 0;k<4;k++)
          assert(EQIV(frho_spline[i][j][k], _h_d_frho_spline[i][j][k]));
  for (int i = 0;i<NGRID;i++)   assert(EQIV(rval[i], _h_d_rval[i]));
  for (int i = 0;i<NGRID;i++)   assert(EQIV(rhoval[i], _h_d_rhoval[i]));
  for (int i = 0;i<size;i++)  for (int j = 0; j<_NNM; j++)
	  assert(EQIV(nindex[i][j], _h_d_nindex[i][j]));
  for (int i = 0;i<size;i++) assert(EQIV(nn[i], _h_d_nn[i]));
  for (int i = 0;i<size;i++) assert(G_Vector3(_SR[i])==G_Vector3(_h_d_SR[i]));
  for (int i = 0;i<size;i++) assert(G_Vector3(_F[i])==G_Vector3(_h_d_F[i]));
  for (int i = 0;i<size;i++) assert(G_Matrix33(_VIRIAL_IND[i])==G_Matrix33(_h_d_VIRIAL_IND[i]));
  for (int i = 0;i<10;i++)   assert(EQIV(fscalars[i], _h_d_fscalars[i]));

  INFO_Printf("I am about to get out of check_host_device memory transfer\n");
  return 0;
}
#endif



void EAMFrame::free_device_ptr() {
  hipFree(_d_rho);
  hipFree(_d_rhop);
  hipFree(_d_phi);
  hipFree(_d_phip);

  hipFree(_d_phix);
  hipFree(_d_phipx);
  hipFree(_d_frho);
  hipFree(_d_frhop);
  hipFree(_d_rho_spline);
  hipFree(_d_phi_spline);
  hipFree(_d_phix_spline);
  hipFree(_d_frho_spline);
  hipFree(_d_rval);
  hipFree(_d_rhoval);

  hipFree(_d_atpe3b);
  hipFree(_d_rhotot);
  hipFree(_d_embf);
  hipFree(_d_embfp);
  hipFree(_d_EPOT);
  hipFree(_d_H_element);
  hipFree(_d_EPOT_IND);
  hipFree(_d_species);        
  hipFree(_d_nindex);
  hipFree(_d_nn);
  hipFree(_d_SR);
  hipFree(_d_F);
  hipFree(_d_F_padding);
  hipFree(_d_VIRIAL_IND_element);
  hipFree(_d_VIRIAL_element);

  hipFree(_d_fscalars);
  Free(fscalars);
}

/* This is a simple test for GPU. run the function to see if maxErro == 0. If not, GPU device is not set correctly */
__global__ void saxpy(int n, float a, const float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}
int EAMFrame::test_saxpy(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  for (int i = 0; i < N; i++) { x[i] = 1.0f; y[i] = 2.0f; }
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = max(maxError, abs(y[i]-4.0f));
  INFO_Printf("Max error: %f\n", maxError);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  return 0;
}

